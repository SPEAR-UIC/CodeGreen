
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <hip/hip_runtime.h>

#include <random>

// Error checking macro to catch CUDA API errors.
#define CUDA_CHECK(call)                                                        \
  do {                                                                          \
    hipError_t err = call;                                                     \
    if (err != hipSuccess) {                                                   \
      fprintf(stderr, "CUDA error in %s (%s:%d): %s\n", #call, __FILE__, __LINE__, hipGetErrorString(err)); \
      exit(EXIT_FAILURE);                                                       \
    }                                                                           \
  } while (0)

template <typename T>
void reference(
    const T* input,
    const T* dense,
    T* output,
    int embedding_dim,
    int batch_size,
    const int* offset)
{
  for (int batch_idx = 0; batch_idx < batch_size; batch_idx++) {
    // cache the offset values
    const int start = offset[batch_idx];
    const int range = offset[batch_idx + 1] - start;
    for (int idx = 0; idx < embedding_dim; idx++) {
      const T dense_elem = dense[batch_idx * embedding_dim + idx];
      for (int nested_idx = idx; nested_idx < range; nested_idx += embedding_dim) {
        output[start + nested_idx] =
          input[start + nested_idx] + dense_elem;
      }
    }
  }
}

template <typename T>
__global__ void dense_esuhm(
    const T* input,
    const T* dense,
          T* output,
    int embedding_dim,
    const int* offset)
{
  const int batch_idx = blockIdx.x; // one batch per block
  const int start = offset[batch_idx];
  const int range = offset[batch_idx + 1] - start;
  const int grain_size = blockDim.x;
  const int tid = threadIdx.x;
  for (int idx = tid; idx < embedding_dim; idx += grain_size) {
    T dense_elem = dense[batch_idx * embedding_dim + idx];
    // Using a pragma to encourage unrolling the inner loop when possible.
    #pragma unroll
    for (int nested_idx = idx; nested_idx < range; nested_idx += embedding_dim) {
      output[start + nested_idx] = input[start + nested_idx] + dense_elem;
    }
  }
}

template <typename T>
__global__ void dense_esuhm2(
    const T* input,
    const T* dense,
          T* output,
    int embedding_dim,
    const int* offset)
{
  const int batch_idx = blockIdx.x;
  const int start = offset[batch_idx];
  const int range = offset[batch_idx + 1] - start;
  for (int idx = threadIdx.x; idx < embedding_dim; idx += blockDim.x) {
    T dense_elem = dense[batch_idx * embedding_dim + idx];
    #pragma unroll
    for (int nested_idx = idx; nested_idx < range; nested_idx += embedding_dim) {
      output[start + nested_idx] = input[start + nested_idx] + dense_elem;
    }
  }
}

int main(int argc, char* argv[])
{
  if (argc != 4) {
    printf("Usage: %s <number of rows> <batch size> <repeat>\n", argv[0]);
    return EXIT_FAILURE;
  }
  const int nrows = atoi(argv[1]);
  const int batch_size = atoi(argv[2]);
  const int repeat = atoi(argv[3]);
  assert(nrows > batch_size * batch_size);

  printf("Number of rows in the embedding table: %d\n", nrows);
  printf("Batch size: %d\n", batch_size);

  // Set device flag for blocking synchronization (which can reduce power consumption on CPU side)
  CUDA_CHECK(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));

  const int embed_dims[] = {768, 2048, 12288};

  for (size_t n = 0; n < sizeof(embed_dims)/sizeof(int); n++) {
    int ncols = embed_dims[n];
    printf("\nEmbedding dimension: %d\n", ncols);

    int input_size = nrows * ncols;  // same as output size
    size_t input_size_bytes = input_size * sizeof(float);

    int dense_size = batch_size * ncols;
    size_t dense_size_bytes = dense_size * sizeof(float);

    int input_offset_bytes = (batch_size + 1) * sizeof(int);

    // Allocate host memory using pinned memory (allows for faster transfers)
    float *input, *dense, *output_k1, *output_k2, *output_ref;
    CUDA_CHECK(hipHostMalloc(&input, input_size_bytes, hipHostMallocDefault)); // embedding table
    CUDA_CHECK(hipHostMalloc(&dense, dense_size_bytes, hipHostMallocDefault)); // dense features for the batch
    CUDA_CHECK(hipHostMalloc(&output_k1, input_size_bytes, hipHostMallocDefault));
    CUDA_CHECK(hipHostMalloc(&output_k2, input_size_bytes, hipHostMallocDefault));
    CUDA_CHECK(hipHostMalloc(&output_ref, input_size_bytes, hipHostMallocDefault));
    int *input_offset;
    CUDA_CHECK(hipHostMalloc(&input_offset, input_offset_bytes, hipHostMallocDefault));

    // Create valid offsets.
    srand(123);
    input_offset[0] = 0;
    for (int i = 1; i <= batch_size; i++)
      input_offset[i] = input_offset[i-1] + (rand() % batch_size + 1) * ncols;

    std::default_random_engine rng(123);
    std::uniform_real_distribution<float> distr(-1.f, 1.f);
    for (int i = 0; i < dense_size; i++) {
      dense[i] = distr(rng);
    }
    for (int i = 0; i < input_size; i++) {
      input[i] = distr(rng);
      output_ref[i] = 0;
    }

    // Compute reference result on host.
    reference(input, dense, output_ref, ncols, batch_size, input_offset);

    // Allocate device memory.
    float *d_input, *d_dense, *d_output;
    CUDA_CHECK(hipMalloc(&d_input, input_size_bytes));
    CUDA_CHECK(hipMalloc(&d_dense, dense_size_bytes));
    CUDA_CHECK(hipMalloc(&d_output, input_size_bytes));

    int* d_input_offset;
    CUDA_CHECK(hipMalloc(&d_input_offset, input_offset_bytes));

    // Transfer data to device.
    CUDA_CHECK(hipMemcpy(d_input, input, input_size_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_dense, dense, dense_size_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_input_offset, input_offset, input_offset_bytes, hipMemcpyHostToDevice));

    // Use CUDA events for timing.
    hipEvent_t event_start, event_stop;
    CUDA_CHECK(hipEventCreate(&event_start));
    CUDA_CHECK(hipEventCreate(&event_stop));

    // Explore different block sizes.
    for (int block_size = 128; block_size <= 1024; block_size *= 2) {
      printf("Block size: %d\n", block_size);

      // First kernel version (dense_esuhm)
      CUDA_CHECK(hipMemset(d_output, 0, input_size_bytes));
      CUDA_CHECK(hipDeviceSynchronize());
      CUDA_CHECK(hipEventRecord(event_start, 0));

      for (int i = 0; i < repeat; i++)
        dense_esuhm<<<batch_size, block_size>>>(d_input, d_dense, d_output, ncols, d_input_offset);
      
      CUDA_CHECK(hipEventRecord(event_stop, 0));
      CUDA_CHECK(hipEventSynchronize(event_stop));
      float time_ms = 0.f;
      CUDA_CHECK(hipEventElapsedTime(&time_ms, event_start, event_stop));
      printf("Average execution time of dense embedding kernel (k1): %f (us)\n", (time_ms * 1e3f) / repeat);
      CUDA_CHECK(hipMemcpy(output_k1, d_output, input_size_bytes, hipMemcpyDeviceToHost));

      // Second kernel version (dense_esuhm2)
      CUDA_CHECK(hipMemset(d_output, 0, input_size_bytes));
      CUDA_CHECK(hipDeviceSynchronize());
      CUDA_CHECK(hipEventRecord(event_start, 0));
      
      for (int i = 0; i < repeat; i++)
        dense_esuhm2<<<batch_size, block_size>>>(d_input, d_dense, d_output, ncols, d_input_offset);
      
      CUDA_CHECK(hipEventRecord(event_stop, 0));
      CUDA_CHECK(hipEventSynchronize(event_stop));
      CUDA_CHECK(hipEventElapsedTime(&time_ms, event_start, event_stop));
      printf("Average execution time of dense embedding kernel (k2): %f (us)\n", (time_ms * 1e3f) / repeat);
      CUDA_CHECK(hipMemcpy(output_k2, d_output, input_size_bytes, hipMemcpyDeviceToHost));

      // Validate results.
      bool ok = true;
      for (int i = 0; i < input_size; i++) {
        if (fabsf(output_k1[i] - output_ref[i]) > 1e-3f ||
            fabsf(output_k2[i] - output_ref[i]) > 1e-3f) {
          ok = false;
          break;
        }
      }
      printf("%s\n", ok ? "PASS" : "FAIL");
    }

    // Cleanup device memory.
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_dense));
    CUDA_CHECK(hipFree(d_output));
    CUDA_CHECK(hipFree(d_input_offset));

    // Cleanup host memory.
    CUDA_CHECK(hipHostFree(input));
    CUDA_CHECK(hipHostFree(dense));
    CUDA_CHECK(hipHostFree(output_k1));
    CUDA_CHECK(hipHostFree(output_k2));
    CUDA_CHECK(hipHostFree(output_ref));
    CUDA_CHECK(hipHostFree(input_offset));

    CUDA_CHECK(hipEventDestroy(event_start));
    CUDA_CHECK(hipEventDestroy(event_stop));
  }

  return EXIT_SUCCESS;
}
