
/*
   Copyright (c) 2015-2016 Advanced Micro Devices, Inc. All rights reserved.

   Permission is hereby granted, free of charge, to any person obtaining a copy
   of this software and associated documentation files (the "Software"), to deal
   in the Software without restriction, including without limitation the rights
   to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
   copies of the Software, and to permit persons to whom the Software is
   furnished to do so, subject to the following conditions:

   The above copyright notice and this permission notice shall be included in
   all copies or substantial portions of the Software.

   THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
   IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
   FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL THE
   AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
   LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
   OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
   THE SOFTWARE.
 */

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <chrono>
#include <hip/hip_runtime.h>

#define MAXDISTANCE    (200)

/**
 * Returns the lesser of the two unsigned integers a and b
 */
unsigned int minimum(unsigned int a, unsigned int b)
{
  return (b < a) ? b : a;
}

/**
 * Reference CPU implementation of FloydWarshall PathFinding
 * for performance comparison
 * @param pathDistanceMatrix Distance between nodes of a graph
 * @param intermediate node between two nodes of a graph
 * @param number of nodes in the graph
 */
void floydWarshallCPUReference(unsigned int * pathDistanceMatrix,
    unsigned int * pathMatrix, unsigned int numNodes)
{
  unsigned int distanceYtoX, distanceYtoK, distanceKtoX, indirectDistance;
  unsigned int width = numNodes;
  unsigned int yXwidth;

  for(unsigned int k = 0; k < numNodes; ++k)
  {
    for(unsigned int y = 0; y < numNodes; ++y)
    {
      yXwidth =  y*numNodes;
      for(unsigned int x = 0; x < numNodes; ++x)
      {
        distanceYtoX = pathDistanceMatrix[yXwidth + x];
        distanceYtoK = pathDistanceMatrix[yXwidth + k];
        distanceKtoX = pathDistanceMatrix[k * width + x];

        indirectDistance = distanceYtoK + distanceKtoX;

        if(indirectDistance < distanceYtoX)
        {
          pathDistanceMatrix[yXwidth + x] = indirectDistance;
          pathMatrix[yXwidth + x]         = k;
        }
      }
    }
  }
}

/*!
 * The floyd Warshall algorithm is a multipass algorithm
 * that calculates the shortest path between each pair of
 * nodes represented by pathDistanceBuffer.
 */
__global__ void floydWarshallPass(
    unsigned int *__restrict__ pathDistanceBuffer,
    unsigned int *__restrict__ pathBuffer,
    const unsigned int numNodes,
    const unsigned int pass)
{
  int xValue = threadIdx.x + blockIdx.x * blockDim.x;
  int yValue = threadIdx.y + blockIdx.y * blockDim.y;

  int k = pass;
  int oldWeight = pathDistanceBuffer[yValue * numNodes + xValue];
  int tempWeight = pathDistanceBuffer[yValue * numNodes + k] +
                   pathDistanceBuffer[k * numNodes + xValue];

  if (tempWeight < oldWeight)
  {
    pathDistanceBuffer[yValue * numNodes + xValue] = tempWeight;
    pathBuffer[yValue * numNodes + xValue] = k;
  }
}

int main(int argc, char** argv) {
  if (argc != 4) {
    printf("Usage: %s <number of nodes> <iterations> <block size>\n", argv[0]);
    return 1;
  }
  // There are three required command-line arguments
  unsigned int numNodes = atoi(argv[1]);
  unsigned int numIterations = atoi(argv[2]);
  unsigned int blockSize = atoi(argv[3]);

  if(numNodes % blockSize != 0) {
    numNodes = (numNodes / blockSize + 1) * blockSize;
  }

  unsigned int* pathMatrix = NULL;
  unsigned int* pathDistanceMatrix = NULL;
  unsigned int* verificationPathDistanceMatrix = NULL;
  unsigned int* verificationPathMatrix = NULL;
  unsigned int matrixSizeBytes = numNodes * numNodes * sizeof(unsigned int);

  pathDistanceMatrix = (unsigned int *) malloc(matrixSizeBytes);
  assert (pathDistanceMatrix != NULL);

  pathMatrix = (unsigned int *) malloc(matrixSizeBytes);
  assert (pathMatrix != NULL);

  srand(2);
  for(unsigned int i = 0; i < numNodes; i++)
    for(unsigned int j = 0; j < numNodes; j++)
    {
      int index = i*numNodes + j;
      pathDistanceMatrix[index] = rand() % (MAXDISTANCE + 1);
    }
  for(unsigned int i = 0; i < numNodes; ++i)
  {
    unsigned int iXWidth = i * numNodes;
    pathDistanceMatrix[iXWidth + i] = 0;
  }

  for(unsigned int i = 0; i < numNodes; ++i)
  {
    for(unsigned int j = 0; j < i; ++j)
    {
      pathMatrix[i * numNodes + j] = i;
      pathMatrix[j * numNodes + i] = j;
    }
    pathMatrix[i * numNodes + i] = i;
  }

  verificationPathDistanceMatrix = (unsigned int *) malloc(numNodes * numNodes * sizeof(int));
  assert (verificationPathDistanceMatrix != NULL);

  verificationPathMatrix = (unsigned int *) malloc(numNodes * numNodes * sizeof(int));
  assert(verificationPathMatrix != NULL);

  memcpy(verificationPathDistanceMatrix, pathDistanceMatrix,
      numNodes * numNodes * sizeof(int));
  memcpy(verificationPathMatrix, pathMatrix, numNodes*numNodes*sizeof(int));

  unsigned int numPasses = numNodes;

  unsigned int globalThreads[2] = {numNodes, numNodes};
  unsigned int localThreads[2] = {blockSize, blockSize};

  if((unsigned int)(localThreads[0] * localThreads[0]) > 256)
  {
    blockSize = 16;
    localThreads[0] = blockSize;
    localThreads[1] = blockSize;
  }

  dim3 grids( globalThreads[0]/localThreads[0], globalThreads[1]/localThreads[1]);
  dim3 threads (localThreads[0],localThreads[1]);

  unsigned int *pathDistanceBuffer, *pathBuffer;
  hipMalloc((void**)&pathDistanceBuffer, matrixSizeBytes);
  hipMalloc((void**)&pathBuffer, matrixSizeBytes);

  // Allocate a device memory buffer to hold the original path distance matrix.
  unsigned int *origPathDistanceBuffer;
  hipMalloc((void**)&origPathDistanceBuffer, matrixSizeBytes);
  // Copy the initial (static) matrix from host to device once.
  hipMemcpy(origPathDistanceBuffer, pathDistanceMatrix, matrixSizeBytes, hipMemcpyHostToDevice);

  float total_time = 0.f;

  // For each iteration, use a fast device-to-device copy to reset the working matrix.
  for (unsigned int n = 0; n < numIterations; n++) {
    // Instead of copying from host to device each iteration, copy from the
    // stored device-resident original matrix to the working buffer.
    hipMemcpy(pathDistanceBuffer, origPathDistanceBuffer, matrixSizeBytes, hipMemcpyDeviceToDevice);

    hipDeviceSynchronize();
    auto start = std::chrono::steady_clock::now();

    for(unsigned int i = 0; i < numPasses; i++)
    {
      floydWarshallPass <<< grids, threads >>> (pathDistanceBuffer, pathBuffer, numNodes, i);
    }

    hipDeviceSynchronize();
    auto end = std::chrono::steady_clock::now();
    auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
    total_time += time;
  }

  printf("Average kernel execution time %f (s)\n", (total_time * 1e-9f) / numIterations);

  hipMemcpy(pathDistanceMatrix, pathDistanceBuffer, matrixSizeBytes, hipMemcpyDeviceToHost);

  hipFree(pathDistanceBuffer);
  hipFree(pathBuffer);
  hipFree(origPathDistanceBuffer);

  // verify
  floydWarshallCPUReference(verificationPathDistanceMatrix, verificationPathMatrix, numNodes);
  if(memcmp(pathDistanceMatrix, verificationPathDistanceMatrix, matrixSizeBytes) == 0)
  {
    printf("PASS\n");
  }
  else
  {
    printf("FAIL\n");
    if (numNodes <= 8)
    {
      for (unsigned int i = 0; i < numNodes; i++) {
        for (unsigned int j = 0; j < numNodes; j++)
          printf("host: %u ", verificationPathDistanceMatrix[i*numNodes+j]);
        printf("\n");
      }
      for (unsigned int i = 0; i < numNodes; i++) {
        for (unsigned int j = 0; j < numNodes; j++)
          printf("device: %u ", pathDistanceMatrix[i*numNodes+j]);
        printf("\n");
      }
    }
  }

  free(pathDistanceMatrix);
  free(pathMatrix);
  free(verificationPathDistanceMatrix);
  free(verificationPathMatrix);
  return 0;
}
